#include "hip/hip_runtime.h"
/*
============================================================================
Author        : G. Barlas
Version       : 1.0
Last modified : December 2014
License       : Released under the GNU GPL 3.0
Description   :
To build use  : make
============================================================================
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "pgm.h"


const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;



void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;   //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  
  *acc = new int[rBins * degreeBins];                  //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros

  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;


  for (int i = 0; i < w; i++)     //por cada pixel
    for (int j = 0; j < h; j++)
      {
        int idx = j * w + i;
        if (pic[idx] > 0)         //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;                       // y-coord has to be reversed

            float theta = 0;                              // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);

                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++;       //+1 para este radio r y este theta

                theta += radInc;
              }
          }
      }
}


//TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }


// declaración de variables en scope global
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

/*
*  params:
*    pic    -> arreglo con pixeles de la imagen
*    w      -> largo
*    h      -> alto
*    acc    -> store de calculos
*    rMax   -> distancia maxima
*    rScale -> escala de distancia
*/
__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int gloID   = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;

          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique

          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }  
}

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID   = (blockIdx.x) * blockDim.x + threadIdx.x;
  
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  //definicion de eventos de CUDA
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //lectura de imagen
  PGMImage inImg (argv[1]);

  //calculo de dimensiones
  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  //secuencial, sirve para comparacion final
  CPU_HoughTran (inImg.pixels, w, h, &cpuht);

  //pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;

  //pre-calculo de los valores para las funciones de seno y coseno
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  //calculo de distancia maxima
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  //copiamos los valores calculados a las variables globales
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  //setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  //alocacion de memoria
  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  //execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  int blockNum = ceil (w * h / 256);

  hipEventRecord(start); //inicio de medicion de tiempo

  //llamada de kernel
  GPU_HoughTranConst <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

  hipEventRecord(stop); //finaliza medicion de tiempo

  //barrera de sincronizacion
  hipDeviceSynchronize();

  //get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  //barrera de sincronizacion
  hipEventSynchronize(stop);

  //comparacion de resultados CPU y GPU
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i] && (cpuht[i] - h_hough[i] > 1))
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }

  //calculo de tiempo transcurrido
  float cons_elapsed = 0;
  hipEventElapsedTime(&cons_elapsed, start, stop);
  printf("Done!\n");

  printf("Elapsed time constant - %f ms.\n", cons_elapsed);

  //clean-up de variables y espacio en memoria
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);

  free(h_hough);
  free(pcCos);
  free(pcSin);

  return 0;
}
